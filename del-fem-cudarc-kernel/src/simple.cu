#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {

__global__
void solve_diffuse_jacobi(
    const uint32_t num_vtx,
    const uint32_t *vtx2idx,
    const uint32_t *idx2vtx,
    float lambda,
    const float *vtx2rhs,
    float *vtx2lhs_ini,
    float *vtx2lhs_upd,
    float *vtx2res)
{
    int i_vtx = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_vtx >= num_vtx) { return; }
    //
    float rhs = vtx2rhs[i_vtx];
    for(uint32_t idx = vtx2idx[i_vtx]; idx < vtx2idx[i_vtx+1]; ++idx ) {
        uint32_t j_vtx = idx2vtx[idx];
        rhs += vtx2lhs_ini[j_vtx];
    }
    const float dtmp = float(vtx2idx[i_vtx+1] - vtx2idx[i_vtx]) + lambda;
    vtx2lhs_upd[i_vtx] = rhs / dtmp;
    vtx2res[i_vtx] = rhs - vtx2lhs_ini[i_vtx] * dtmp;
}

}

