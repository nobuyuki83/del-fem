#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {

__global__
void solve_diffuse_jacobi(
    const uint32_t num_vtx,
    const uint32_t *vtx2idx,
    const uint32_t *idx2vtx,
    float lambda,
    const float *vtx2rhs,
    float *vtx2lhs_ini,
    float *vtx2lhs_upd,
    float *vtx2res)
{
    int i_vtx = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_vtx >= num_vtx) { return; }
    //
    float rhs = vtx2rhs[i_vtx];
    for(uint32_t idx = vtx2idx[i_vtx]; idx < vtx2idx[i_vtx+1]; ++idx ) {
        uint32_t j_vtx = idx2vtx[idx];
        rhs += vtx2lhs_ini[j_vtx];
    }
    const float dtmp = float(vtx2idx[i_vtx+1] - vtx2idx[i_vtx]) + lambda;
    vtx2lhs_upd[i_vtx] = rhs / dtmp;
    vtx2res[i_vtx] = rhs - vtx2lhs_ini[i_vtx] * dtmp;
}

__global__
void solve_diffuse3_jacobi(
    const uint32_t num_vtx,
    const uint32_t *vtx2idx,
    const uint32_t *idx2vtx,
    float lambda,
    const float *vtx2rhs,
    float *vtx2lhs_ini,
    float *vtx2lhs_upd,
    float *vtx2res)
{
    int i_vtx = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_vtx >= num_vtx) { return; }
    //
    float rhs[3] = {
        vtx2rhs[i_vtx*3+0],
        vtx2rhs[i_vtx*3+1],
        vtx2rhs[i_vtx*3+2] };
    for(uint32_t idx = vtx2idx[i_vtx]; idx < vtx2idx[i_vtx+1]; ++idx ) {
        uint32_t j_vtx = idx2vtx[idx];
        rhs[0] += vtx2lhs_ini[j_vtx*3+0];
        rhs[1] += vtx2lhs_ini[j_vtx*3+1];
        rhs[2] += vtx2lhs_ini[j_vtx*3+2];
    }
    const float dtmp = float(vtx2idx[i_vtx+1] - vtx2idx[i_vtx]) + lambda;
    vtx2lhs_upd[i_vtx*3+0] = rhs[0] / dtmp;
    vtx2lhs_upd[i_vtx*3+1] = rhs[1] / dtmp;
    vtx2lhs_upd[i_vtx*3+2] = rhs[2] / dtmp;
    float r0 = rhs[0] - vtx2lhs_ini[i_vtx*3+0] * dtmp;
    float r1 = rhs[1] - vtx2lhs_ini[i_vtx*3+1] * dtmp;
    float r2 = rhs[2] - vtx2lhs_ini[i_vtx*3+2] * dtmp;
    vtx2res[i_vtx] = r0*r0 + r1*r1 + r2*r2;
}

}

