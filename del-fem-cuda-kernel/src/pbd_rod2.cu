#include "hip/hip_runtime.h"
#include <cstdint> // for uint32_t
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "edge2.h"
#include "spring2.h"

extern "C" {

__global__
void solve(
    const uint32_t num_example,
    const uint32_t num_point,
    const float *pnt2xyini,
    const float *pnt2massinv,
    float dt,
    float *gravity,
    float *example2pnt2xydef,
    float *example2pnt2xynew,
    float *example2pnt2velo)
{
    int i_example = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_example >= num_example) { return; }
    // ---------------
    float* pnt2xydef = example2pnt2xydef + i_example * num_point * 2;
    float* pnt2xynew = example2pnt2xynew + i_example * num_point * 2;
    float* pnt2velo = example2pnt2velo + i_example * num_point * 2;
    for(int i_point=0;i_point<num_point;++i_point) {
      pnt2xydef[i_point*2+0] = pnt2xyini[i_point*2+0];
      pnt2xydef[i_point*2+1] = pnt2xyini[i_point*2+1];
      pnt2velo[i_point*2+0] = 0.f;
      pnt2velo[i_point*2+1] = 0.f;
    }
    for(int i_step = 0; i_step<1000; ++i_step) {
        for(int i_point = 0; i_point < num_point; ++i_point) {
            if( pnt2massinv[i_point] == 0.f ){
                continue;
            }
            pnt2xynew[i_point * 2 + 0] =
                pnt2xydef[i_point * 2 + 0] + dt * dt * gravity[0] + dt * pnt2velo[i_point * 2 + 0];
            pnt2xynew[i_point * 2 + 1] =
                pnt2xydef[i_point * 2 + 1] + dt * dt * gravity[1] + dt * pnt2velo[i_point * 2 + 1];
        }
        for(int i_seg = 0; i_seg < num_point - 1; ++i_seg) {
            int ip0 = i_seg;
            int ip1 = i_seg + 1;
            const float* p0_def = pnt2xynew + ip0 * 2;
            const float* p1_def = pnt2xynew + ip1 * 2;
            const float* p0_ini = pnt2xyini + ip0 * 2;
            const float* p1_ini = pnt2xyini + ip1 * 2;
            const float w0 = pnt2massinv[ip0];
            const float w1 = pnt2massinv[ip1];
            const float len_ini = edge2::length(p0_ini, p1_ini);
            const spring2::ReturnPbd r = spring2::pbd(p0_def, p1_def, len_ini, w0, w1);
            pnt2xynew[ip0 * 2 + 0] += r.dp0[0];
            pnt2xynew[ip0 * 2 + 1] += r.dp0[1];
            pnt2xynew[ip1 * 2 + 0] += r.dp1[0];
            pnt2xynew[ip1 * 2 + 1] += r.dp1[1];
        }
        for(int i_point = 0; i_point < num_point; ++i_point) {
            pnt2velo[i_point] = (pnt2xynew[i_point] - pnt2xydef[i_point]) / dt;
            pnt2xydef[i_point] = pnt2xynew[i_point];
        }
    }
}

}